#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Element::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&in1Tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&in2Tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outTensor));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // set descriptors
  helperSetBroadcastableTensorDescriptor(inputs[0], outputs[0], in1Tensor);
  helperSetBroadcastableTensorDescriptor(inputs[1], outputs[0], in2Tensor);
  helperSetTensorDescriptor(outputs[0], outTensor);

  hipdnnOpTensorOp_t opType;
  switch (type) {
    case OP_EW_ADD:
      opType = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      opType = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
      HIPDNN_NOT_PROPAGATE_NAN));
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Element::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(in1Tensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(in2Tensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outTensor));
  checkCUDNN(hipdnnDestroyOpTensorDescriptor(opDesc));
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Element::forward(bool block)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  checkCUDNN(hipdnnOpTensor(model->dnn, opDesc, &alpha, in1Tensor, inputs[0].data_ptr,
      &alpha, in2Tensor, inputs[1].data_ptr, &beta, outTensor, outputs[0].data_ptr));
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_element_cost(Element* ele)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  helperSetBroadcastableTensorDescriptor(ele->inputs[0],
      ele->outputs[0], inputTensor);
  helperSetBroadcastableTensorDescriptor(ele->inputs[1],
      ele->outputs[0], biasTensor);
  helperSetTensorDescriptor(ele->outputs[0], outputTensor);
  //int inputN = ele->inputs[0].dim[0];
  //int inputC = max(ele->inputs[0].dim[1], 1);
  //int inputH = max(ele->inputs[0].dim[2], 1);
  //int inputW = max(ele->inputs[0].dim[3], 1);
  //checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
  //    HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));

  hipdnnOpTensorOp_t opType;
  switch (ele->type) {
    case OP_EW_ADD:
      opType = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      opType = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
      HIPDNN_NOT_PROPAGATE_NAN));

  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    checkCUDNN(hipdnnOpTensor(dnn, opDesc, &alpha, inputTensor, inputPtr,
        &alpha, biasTensor, filterPtr, &beta, outputTensor, outputPtr));
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  ele->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Element]: i(%d %d %d %d) type(%d) cost(%.4lf)\n",
           ele->inputs[0].dim[0], ele->inputs[0].dim[1], ele->inputs[0].dim[2],
           ele->inputs[0].dim[3], ele->type, ele->runtime);
}

